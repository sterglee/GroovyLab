#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <memory>
#include <iostream>

#include "CUDAOps_KernelOps.h"

#include <hipfft/hipfft.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>


  // Thread block size 
#define BLOCK_SIZE 16
 

#define SDATA( index)      cutilBankChecker(sdata, index)


inline bool IsGPUCapableP2P(hipDeviceProp_t *pProp)
{
#ifdef _WIN32
    return (bool)(pProp->tccDriver ? true : false);
#else
    return (bool)(pProp->major >= 2);
#endif
}

inline bool IsAppBuiltAs64()
{
#if defined(__x86_64) || defined(AMD64) || defined(_M_AMD64)
    return 1;
#else
    return 0;
#endif
}


void cudafft( hipfftReal * hinData, int NX, float *hOut ) {
/*  hipfftHandle plan;
  hipfftComplex *dfftdata;
  
  hipMalloc( (void **)&dfftdata, sizeof(hipfftComplex)*(NX/2+1));
  if (hipGetLastError() != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return;
    }
  
  // Copy input float array to hipfftComplex device array 
 	hipMemcpy(dfftdata,  hinData,  NX*sizeof(float), hipMemcpyHostToDevice);
  
 	if (hipfftPlan1d(&plan, NX, HIPFFT_R2C, 1) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
   }
  
   // use the CUFFT plan to transform the signal in place
   if (hipfftExecR2C(plan,  (hipfftReal *) dfftdata, dfftdata ) != HIPFFT_SUCCESS) {
   fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
   return;
   }
   
   if (hipDeviceSynchronize() != hipSuccess) {
     fprintf(stderr, "Cuda error: Failed to synchronize \n");
     return;
  
  }

// copy computed results in device space to host
       hipMemcpy( hOut, dfftdata,  NX*sizeof(float), hipMemcpyDeviceToHost);
	   
    hipfftDestroy(plan);
*/  
  }
     

	 JNIEXPORT void JNICALL Java_CUDAOps_KernelOps_cudafft
  (JNIEnv *env, jobject obj, jfloatArray inData, jint N, jfloatArray outRealImsFFT)
{
    jfloat *data = env->GetFloatArrayElements(inData, 0);
    jfloat *outRealIms = env->GetFloatArrayElements(outRealImsFFT, 0);
    
	// perform the FFT
   cudafft( data, N, outRealIms );
   
    env->ReleaseFloatArrayElements( inData, data, 0);
    env->ReleaseFloatArrayElements( outRealImsFFT, outRealIms, 0);
    
	}

static int  simple_sgemm(const float *h_A, int hA, int wA,
	const float *h_B, int wB,  float *h_C) {
  /*  int hB = wA; 
    int hC = hA;  int wC = wB;
    float alpha = 1.0f;
    float beta = 0.0f;
    float *d_A = 0;
    float *d_B = 0;
    float *d_C = 0;
    hipblasHandle_t handle;
    hipblasStatus_t status;
    

    // Initialize CUBLAS 
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    // Allocate device memory for the matrices 
    if (hipMalloc((void **)&d_A, hA * wA * sizeof(d_A[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc((void **)&d_B,  hB * wB * sizeof(d_B[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc((void **)&d_C, hC * wC * sizeof(d_C[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
        return EXIT_FAILURE;
    }

    // Initialize the device matrices with the host matrices 
    hipMemcpy( d_A, h_A, hA*wA*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_B, h_B, hB*wB*sizeof(float), hipMemcpyHostToDevice);
    
    int lda = hA;
    int ldb = hB;
    int ldc = hC;
 // Performs operation using cublas 
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, hA, wB,  wA, &alpha, 
    	    d_A, lda, d_B, ldb, &beta, d_C, ldc);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }
    
    hipMemcpy( h_C, d_C, hC*wC*sizeof(float), hipMemcpyDeviceToHost);

 
    // Memory clean up 
    if (hipFree(d_A) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_B) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (B)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_C) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (C)\n");
        return EXIT_FAILURE;
    }

    // Shutdown 
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
	*/
return 1;
}



static int  simple_dgemm(const double *h_A, int hA, int wA,
	const double *h_B, int wB,  double *h_C) {
 /*   int hB = wA; 
    int hC = hA;
    int wC = wB;
    double alpha = 1.0;
    double beta = 0.0;
    double *d_A = 0;
    double *d_B = 0;
    double *d_C = 0;
    hipblasHandle_t handle;
    hipblasStatus_t status;
    

    // Initialize CUBLAS 
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    // Allocate device memory for the matrices 
    if (hipMalloc((void **)&d_A, hA * wA * sizeof(d_A[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc((void **)&d_B,  hB * wB * sizeof(d_B[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc((void **)&d_C, hC * wC * sizeof(d_C[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
        return EXIT_FAILURE;
    }

    // Initialize the device matrices with the host matrices 
    hipMemcpy( d_A, h_A, hA*wA*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy( d_B, h_B, hB*wB*sizeof(double), hipMemcpyHostToDevice);
    
    int lda = hA;
    int ldb = hB;
    int ldc = hC;
 // Performs operation using cublas 
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, hA, wB,  wA, &alpha, 
    	    d_A, lda, d_B, ldb, &beta, d_C, ldc);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }
    
    hipMemcpy( h_C, d_C, hC*wC*sizeof(double), hipMemcpyDeviceToHost);

 
    // Memory clean up 
    if (hipFree(d_A) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_B) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (B)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_C) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (C)\n");
        return EXIT_FAILURE;
    }

    // Shutdown 
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
	*/
return 1;
}


    
__global__ void add_matrix(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] + b[idx];
}


__global__ void dadd_matrix(double *a, double *b, double *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] + b[idx];
}

__global__ void subtract_matrix(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] - b[idx];
}

__global__ void dsubtract_matrix(double *a, double *b, double *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] - b[idx];
}

__global__ void dmul_Scalar_matrix(double *a, double value, double *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx]*value;
}
	

__global__ void mul_Scalar_matrix(float *a, float value, float *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx]*value;
}

void cuda_matrixAdd(float *a_h, float *b_h, float *c_h, int N)
{
    float *a_d, *b_d, *c_d;
   
    size_t size = N * sizeof (float);
        
    // allocate memory in the GPU device for a, b and c
    hipMalloc((void **) & a_d, size);
    hipMalloc((void **) & b_d, size);
    hipMalloc((void **) & c_d, size);
    // copy from host to GPU device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    // do calculations on device
    int block_size = 256;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
    add_matrix <<<n_blocks, block_size >>>(a_d, b_d, c_d, N);
    // Retrieve results from the device
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
        // Cleanup
        
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    
}


void dcuda_matrixAdd(double *a_h, double *b_h, double *c_h, int N)
{
    double *a_d, *b_d, *c_d;
   
    size_t size = N * sizeof (double);
        
    // allocate memory in the GPU device for a, b and c
    hipMalloc((void **) & a_d, size);
    hipMalloc((void **) & b_d, size);
    hipMalloc((void **) & c_d, size);
    // copy from host to GPU device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    // do calculations on device
    int block_size = 256;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
    dadd_matrix <<<n_blocks, block_size >>>(a_d, b_d, c_d, N);
    // Retrieve results from the device
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
        // Cleanup
        
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    
}


void cuda_matrixSubtract(float *a_h, float *b_h, float *c_h, int N)
{
    float *a_d, *b_d, *c_d;
   
    size_t size = N * sizeof (float);
        
    // allocate memory in the GPU device for a, b and c
    hipMalloc((void **) & a_d, size);
    hipMalloc((void **) & b_d, size);
    hipMalloc((void **) & c_d, size);
    // copy from host to GPU device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    // do calculations on device
    int block_size = 256;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
    subtract_matrix <<<n_blocks, block_size >>>(a_d, b_d, c_d, N);
    // Retrieve results from the device
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
        // Cleanup
        
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    
}


void dcuda_matrixSubtract(double *a_h, double *b_h, double *c_h, int N)
{
    double *a_d, *b_d, *c_d;
   
    size_t size = N * sizeof (double);
        
    // allocate memory in the GPU device for a, b and c
    hipMalloc((void **) & a_d, size);
    hipMalloc((void **) & b_d, size);
    hipMalloc((void **) & c_d, size);
    // copy from host to GPU device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    // do calculations on device
    int block_size = 256;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
    dsubtract_matrix <<<n_blocks, block_size >>>(a_d, b_d, c_d, N);
    // Retrieve results from the device
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
        // Cleanup
        
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    
}
  
void cuda_matrixMulScalar(float *a_h, float scalarValue, float *c_h, int N)
{
    float *a_d,  *c_d;
   
    size_t size = N * sizeof (float);
        
    // allocate memory in the GPU device for a, b and c
    hipMalloc((void **) & a_d, size);
    hipMalloc((void **) & c_d, size);
    // copy from host to GPU device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    // do calculations on device
    int block_size = 256;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);

  mul_Scalar_matrix  <<<n_blocks, block_size >>>(a_d, scalarValue, c_d, N);
    // Retrieve results from the device
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
        // Cleanup
        
    hipFree(a_d);
    hipFree(c_d);
    
}


void dcuda_matrixMulScalar(double *a_h, double scalarValue, double *c_h, int N)
{
    double *a_d,  *c_d;
   
    size_t size = N * sizeof (double);
        
    // allocate memory in the GPU device for a, b and c
    hipMalloc((void **) & a_d, size);
    hipMalloc((void **) & c_d, size);
    // copy from host to GPU device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    // do calculations on device
    int block_size = 256;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);

  dmul_Scalar_matrix  <<<n_blocks, block_size >>>(a_d, scalarValue, c_d, N);
    // Retrieve results from the device
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
        // Cleanup
        
    hipFree(a_d);
    hipFree(c_d);
    
}
 

// Device multiplication function called by Mul() 
// Compute C = A * B
//   hA is the height of A (i.e. # rows) 
//   wA is the width of A (i.e. # columns)
//   wB is the width of B 
__global__ void Muld(float* A, float* B, int hA, int wA, int wC, float* C) 
{ 
   // each thread computes one element of C
   // by accumulating results into Cvalue
     float Cvalue = 0.0;
	 int row = blockIdx.y * blockDim.y + threadIdx.y;
	 int col = blockIdx.x * blockDim.x + threadIdx.x;
	   
	 if (row >= hA || col >= wC) return;
	    
	 for (int e=0; e<wA; ++e)
	  Cvalue += (A[row*wA+e]) *(B[e*wC+col]);
     
	 C[row*wC+col] = Cvalue;
 } 
	 
	  

// Device multiplication function called by Mul() 
// Compute C = A * B
//   hA is the height of A (i.e. # rows) 
//   wA is the width of A (i.e. # columns)
//   wB is the width of B 
__global__ void dMuld(double* A, double* B, int hA, int wA, int wC, double* C) 
{ 
   // each thread computes one element of C
   // by accumulating results into Cvalue
     	 double Cvalue = 0.0;
	 int row = blockIdx.y * blockDim.y + threadIdx.y;
	 int col = blockIdx.x * blockDim.x + threadIdx.x;
	   
	 if (row >= hA || col >= wC) return;
	    
	 for (int e=0; e<wA; ++e)
	  Cvalue += (A[row*wA+e]) *(B[e*wC+col]);
     
	 C[row*wC+col] = Cvalue;
 } 
	 
	  
	      
 
 
// Host multiplication function 
// Compute C = A * B 
//   hA is the height of A (i.e. # rows)
//   wA is the width of A (i.e. # cols)
//   wB is the width of B 
void Mul(const float* A, const float* B, int hA, int wA, int wB, float* C) 
{ 
    int size; 
 
    // Load A and B to the device 
    float* Ad; 
    size = hA * wA * sizeof(float); 
    hipError_t err = hipMalloc((void**)&Ad, size);
    //printf("CUDA malloc A: %s \n", hipGetErrorString(err));
    err = hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    //printf("Copy A to device: %s \n", hipGetErrorString(err));
    
    float* Bd; 
    int hB = wA;   // #rows of B == #columns of A 
    size = hB * wB * sizeof(float); 
    err = hipMalloc((void**)&Bd, size); 
   // printf("CUDA malloc B: %s \n", hipGetErrorString(err));
    err = hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    //printf("Copy B to device: %s \n", hipGetErrorString(err));
 
    // Allocate C on the device  
    float* Cd; 
    int hC = hA;   // #rows of C == #rows of A
    int wC = wB;   // #columns of C == #columns of B
    size = hC * wC * sizeof(float);
    err = hipMalloc((void**)&Cd, size); 
   // printf("CUDA malloc C: %s \n", hipGetErrorString(err));
    
    // Compute the execution configuration assuming 
    // the matrix dimensions are multiples of BLOCK_SIZE 
    
    /******************** 
    calculates the execution configuration
    effectively the kernel function <Muld> will be
    executed concurrently by BLOCK_SIZE^2 GPU threads
    ************************/
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid((wB + dimBlock.x-1)/dimBlock.x, (hA+dimBlock.y-1) / dimBlock.y); 
    // Launch the device computation 
    Muld<<<dimGrid, dimBlock>>>(Ad, Bd, hA, wA, wC, Cd); 

    err = hipDeviceSynchronize();
    //printf("Run kernel:   %s \n", hipGetErrorString(err));
    
    
    // Read C from the device 
    err = hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
   // printf("Copy C off the device:  %s \n", hipGetErrorString(err));
    
 
    // Free device memory 
    hipFree(Ad); 
    hipFree(Bd); 
    hipFree(Cd); 
}      


 
// Host multiplication function 
// Compute C = A * B 
//   hA is the height of A (i.e. # rows)
//   wA is the width of A (i.e. # cols)
//   wB is the width of B 
void dMul(const double* A, const double* B, int hA, int wA, int wB, double* C) 
{ 
    int size; 
 
    // Load A and B to the device 
    double* Ad; 
    size = hA * wA * sizeof(double); 
    hipError_t err = hipMalloc((void**)&Ad, size);
    //printf("CUDA malloc A: %s \n", hipGetErrorString(err));
    err = hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    //printf("Copy A to device: %s \n", hipGetErrorString(err));
    
    double* Bd; 
    int hB = wA;   // #rows of B == #columns of A 
    size = hB * wB * sizeof(double); 
    err = hipMalloc((void**)&Bd, size); 
   // printf("CUDA malloc B: %s \n", hipGetErrorString(err));
    err = hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    //printf("Copy B to device: %s \n", hipGetErrorString(err));
 
    // Allocate C on the device  
    double* Cd; 
    int hC = hA;   // #rows of C == #rows of A
    int wC = wB;   // #columns of C == #columns of B
    size = hC * wC * sizeof(double);
    err = hipMalloc((void**)&Cd, size); 
   // printf("CUDA malloc C: %s \n", hipGetErrorString(err));
    
    // Compute the execution configuration assuming 
    // the matrix dimensions are multiples of BLOCK_SIZE 
    
    /******************** 
    calculates the execution configuration
    effectively the kernel function <Muld> will be
    executed concurrently by BLOCK_SIZE^2 GPU threads
    ************************/
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid((wB + dimBlock.x-1)/dimBlock.x, (hA+dimBlock.y-1) / dimBlock.y); 
    // Launch the device computation 
    dMuld<<<dimGrid, dimBlock>>>(Ad, Bd, hA, wA, wC, Cd); 

    err = hipDeviceSynchronize();
    //printf("Run kernel:   %s \n", hipGetErrorString(err));
    
    
    // Read C from the device 
    err = hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
   // printf("Copy C off the device:  %s \n", hipGetErrorString(err));
    
 
    // Free device memory 
    hipFree(Ad); 
    hipFree(Bd); 
    hipFree(Cd); 
}      


// add matrices
extern "C"
JNIEXPORT void JNICALL Java_CUDAOps_KernelOps_cma(JNIEnv *env, jobject obj, jfloatArray aArray, jfloatArray bArray, jfloatArray cArray)
{
    
    jfloat *a = env->GetFloatArrayElements( aArray, 0);
    jfloat *b = env->GetFloatArrayElements( bArray, 0);
    jfloat *c = env->GetFloatArrayElements( cArray, 0);
    
    jsize N = env->GetArrayLength( aArray);
    
    cuda_matrixAdd(a, b, c, N);
    
    env->ReleaseFloatArrayElements( aArray, a, 0);
    env->ReleaseFloatArrayElements( bArray, b, 0);
    env->ReleaseFloatArrayElements( cArray, c, 0);
  
}
   

// add matrices
extern "C"
JNIEXPORT void JNICALL Java_CUDAOps_KernelOps_cmad(JNIEnv *env, jobject obj, jdoubleArray aArray, jdoubleArray bArray, jdoubleArray cArray)
{
    
    jdouble *a = env->GetDoubleArrayElements( aArray, 0);
    jdouble *b = env->GetDoubleArrayElements( bArray, 0);
    jdouble *c = env->GetDoubleArrayElements( cArray, 0);
    
    jsize N = env->GetArrayLength( aArray);
    
    dcuda_matrixAdd(a, b, c, N);
    
    env->ReleaseDoubleArrayElements( aArray, a, 0);
    env->ReleaseDoubleArrayElements( bArray, b, 0);
    env->ReleaseDoubleArrayElements( cArray, c, 0);
  
}
   


// multiply with a scalar
extern "C"
JNIEXPORT void JNICALL Java_CUDAOps_KernelOps_cmscalar(JNIEnv *env, jobject obj, jfloatArray aArray, jfloat value, jfloatArray cArray)
{
    
    jfloat  *a = env->GetFloatArrayElements( aArray, 0);
    jfloat  *c = env->GetFloatArrayElements( cArray, 0);
    
    jsize N = env->GetArrayLength( aArray);
    
    cuda_matrixMulScalar(a, value, c, N);
    
    env->ReleaseFloatArrayElements( aArray, a, 0);
    env->ReleaseFloatArrayElements( cArray, c, 0);
}

// multiply with a scalar
extern "C"
JNIEXPORT void JNICALL Java_CUDAOps_KernelOps_cmscalard(JNIEnv *env, jobject obj, jdoubleArray aArray, jdouble value, jdoubleArray cArray)
{
    
    jdouble  *a = env->GetDoubleArrayElements( aArray, 0);
    jdouble  *c = env->GetDoubleArrayElements( cArray, 0);
    
    jsize N = env->GetArrayLength( aArray);
    
    dcuda_matrixMulScalar(a, value, c, N);
    
    env->ReleaseDoubleArrayElements( aArray, a, 0);
    env->ReleaseDoubleArrayElements( cArray, c, 0);
}

  // subtract matrices
extern "C"
JNIEXPORT void JNICALL Java_CUDAOps_KernelOps_cms(JNIEnv *env, jobject obj, jfloatArray aArray, jfloatArray bArray, jfloatArray cArray)
{
    
    jfloat *a = env->GetFloatArrayElements( aArray, 0);
    jfloat *b = env->GetFloatArrayElements( bArray, 0);
    jfloat *c = env->GetFloatArrayElements( cArray, 0);
    
    jsize N = env->GetArrayLength( aArray);
    
    cuda_matrixSubtract(a, b, c, N);
    
    env->ReleaseFloatArrayElements( aArray, a, 0);
    env->ReleaseFloatArrayElements( bArray, b, 0);
    env->ReleaseFloatArrayElements( cArray, c, 0);
}


  // subtract matrices
extern "C"
JNIEXPORT void JNICALL Java_CUDAOps_KernelOps_cmsd(JNIEnv *env, jobject obj, jdoubleArray aArray, jdoubleArray bArray, jdoubleArray cArray)
{
    
    jdouble *a = env->GetDoubleArrayElements( aArray, 0);
    jdouble *b = env->GetDoubleArrayElements( bArray, 0);
    jdouble *c = env->GetDoubleArrayElements( cArray, 0);
    
    jsize N = env->GetArrayLength( aArray);
    
    dcuda_matrixSubtract(a, b, c, N);
    
    env->ReleaseDoubleArrayElements( aArray, a, 0);
    env->ReleaseDoubleArrayElements( bArray, b, 0);
    env->ReleaseDoubleArrayElements( cArray, c, 0);
  
}


// multiply matrices
extern "C"
JNIEXPORT void JNICALL Java_CUDAOps_KernelOps_cmm(JNIEnv *env, jobject obj, jfloatArray aArray, jfloatArray bArray, jfloatArray cArray, jint hA, jint wA, jint wB)
  {
    jfloat *a = env->GetFloatArrayElements( aArray, 0);
    jfloat *b = env->GetFloatArrayElements( bArray, 0);
    jfloat *c = env->GetFloatArrayElements( cArray, 0);
    
	// call the C multiplication routine 
	Mul(a,  b, hA, wA, wB, c); 
	 
	env->ReleaseFloatArrayElements( aArray, a, 0);
	env->ReleaseFloatArrayElements( bArray, b, 0);
	env->ReleaseFloatArrayElements( cArray, c, 0);
	
	}


	
// multiply matrices
extern "C"
JNIEXPORT void JNICALL Java_CUDAOps_KernelOps_cmmd(JNIEnv *env, jobject obj, jdoubleArray aArray, jdoubleArray bArray, jdoubleArray cArray, jint hA, jint wA, jint wB)
  {
    jdouble *a = env->GetDoubleArrayElements( aArray, 0);
    jdouble *b = env->GetDoubleArrayElements( bArray, 0);
    jdouble *c = env->GetDoubleArrayElements( cArray, 0);
    
	// call the C multiplication routine 
	dMul(a,  b, hA, wA, wB, c); 
	 
	env->ReleaseDoubleArrayElements( aArray, a, 0);
	env->ReleaseDoubleArrayElements( bArray, b, 0);
	env->ReleaseDoubleArrayElements( cArray, c, 0);
	
	} 

extern "C"
 JNIEXPORT jint JNICALL Java_CUDAOps_KernelOps_sgemm
  (JNIEnv *env, jobject obj, jfloatArray h_a, jint hA, jint wA, jfloatArray h_b, jint wB, jfloatArray h_c)
  {
    jfloat *ha = env->GetFloatArrayElements(h_a, 0);
    jfloat *hb = env->GetFloatArrayElements(h_b, 0);
    jfloat *hc = env->GetFloatArrayElements(h_c, 0);
    
    simple_sgemm(ha,  hA, wA, hb, wB, hc);
  
    env->ReleaseFloatArrayElements( h_a, ha, 0);
    env->ReleaseFloatArrayElements( h_b, hb, 0);
    env->ReleaseFloatArrayElements( h_c, hc, 0);
	
    return 0;
  }

extern "C"
JNIEXPORT jint JNICALL Java_CUDAOps_KernelOps_dgemm
  (JNIEnv *env, jobject obj, jdoubleArray h_a, jint hA, jint wA, jdoubleArray h_b, jint wB, jdoubleArray h_c) 
   {
    jdouble *ha = env->GetDoubleArrayElements(h_a, 0);
    jdouble *hb = env->GetDoubleArrayElements(h_b, 0);
    jdouble *hc = env->GetDoubleArrayElements(h_c, 0);
    
    simple_dgemm(ha,  hA, wA, hb, wB, hc);
  
    env->ReleaseDoubleArrayElements( h_a, ha, 0);
    env->ReleaseDoubleArrayElements( h_b, hb, 0);
    env->ReleaseDoubleArrayElements( h_c, hc, 0);
	
    return 0;
  }
 
 
extern "C"
 JNIEXPORT jstring JNICALL Java_CUDAOps_KernelOps_getCUDADeviceInfo
  (JNIEnv *env, jobject obj )

{
      FILE * pFile = fopen("temp.txt", "w"); 
      jstring ret;
	  
	  char *s = "Fail";
	  
      fprintf(pFile, " CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");
	  
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

	
    if (error_id != hipSuccess)
    {
        
		fprintf(pFile,"hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		

 	   ret = env->NewStringUTF(s);

       return ret;	
       
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        fprintf(pFile, "There are no available device(s) that support CUDA\n");
		
    }
    else
    {
        fprintf(pFile, "Detected %d CUDA Capable device(s)\n", deviceCount);
		
	}

    int dev, driverVersion = 0, runtimeVersion = 0;

    for (dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        fprintf(pFile, "\nDevice %d: \"%s\"\n", dev, deviceProp.name);
		
		
        // Console log
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
        fprintf(pFile, "  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
		fprintf(pFile, "  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);
		
        char msg[256];
        sprintf(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
        fprintf(pFile, "%s", msg);
		

        fprintf(pFile, "  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
               deviceProp.multiProcessorCount,
               _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
               _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
					   
        fprintf(pFile, "  GPU Clock rate:                                %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

		

#if CUDART_VERSION >= 5000
        // This is supported in CUDA 5.0 (runtime API device properties)
        fprintf(pFile, "  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
		
        fprintf(pFile, "  Memory Bus Width:                              %d-bit\n",   deviceProp.memoryBusWidth);
		
        if (deviceProp.l2CacheSize)
        {
            fprintf(pFile, "  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
			
		}
#else
        // This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
        int memoryClock;
        hipDeviceGetAttribute(&memoryClock, hipDeviceAttributeMemoryClockRate, dev);
        fprintf(pFile, "  Memory Clock rate:                             %.0f Mhz\n", memoryClock * 1e-3f);
		
        int memBusWidth;
        hipDeviceGetAttribute(&memBusWidth, hipDeviceAttributeMemoryBusWidth, dev);
        fprintf(pFile, "  Memory Bus Width:                              %d-bit\n", memBusWidth);
		
        int L2CacheSize;
        hipDeviceGetAttribute(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);

        if (L2CacheSize)
        {
            fprintf(pFile, "  L2 Cache Size:                                 %d bytes\n", L2CacheSize);
			
        }
#endif

        fprintf(pFile, "  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
               deviceProp.maxTexture1D   , deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
               deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
        				
        fprintf(pFile, "  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
               deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
        
		fprintf(pFile, "  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
			   deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);
		

        fprintf(pFile, "  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
		
        fprintf(pFile, "  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
		
        fprintf(pFile, "  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
		
        fprintf(pFile, "  Warp size:                                     %d\n", deviceProp.warpSize);
		
        fprintf(pFile, "  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
		
        fprintf(pFile, "  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
		
        fprintf(pFile, "  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
		
        fprintf(pFile, "  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
			   
        fprintf(pFile, "  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
    	
		fprintf(pFile, "  Texture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
        
		fprintf(pFile, "  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
        
		fprintf(pFile, "  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
        
		fprintf(pFile, "  Integrated GPU sharing Host Memory:            %s\n", deviceProp.integrated ? "Yes" : "No");
        
		fprintf(pFile, "  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
        
		fprintf(pFile, "  Alignment requirement for Surfaces:            %s\n", deviceProp.surfaceAlignment ? "Yes" : "No");
        
		fprintf(pFile, "  Device has ECC support:                        %s\n", deviceProp.ECCEnabled ? "Enabled" : "Disabled");
#ifdef WIN32
        
		fprintf(pFile, "  CUDA Device Driver Mode (TCC or WDDM):         %s\n", deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif
        
		fprintf(pFile, "  Device supports Unified Addressing (UVA):      %s\n", deviceProp.unifiedAddressing ? "Yes" : "No");
        
		fprintf(pFile, "  Device PCI Bus ID / PCI location ID:           %d / %d\n", deviceProp.pciBusID, deviceProp.pciDeviceID);
		
		
        const char *sComputeMode[] =
        {
            "Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
            "Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
            "Prohibited (no host thread can use ::hipSetDevice() with this device)",
            "Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
            "Unknown",
            NULL
        };
        fprintf(pFile, "  Compute Mode:\n");
		
        fprintf(pFile, "     < %s >\n", sComputeMode[deviceProp.computeMode]);
		
    }

    // If there are 2 or more GPUs, query to determine whether RDMA is supported
    if (deviceCount >= 2)
    {
        hipDeviceProp_t prop[64];
        int gpuid[64]; // we want to find the first two GPU's that can support P2P
        int gpu_p2p_count = 0;

        for (int i=0; i < deviceCount; i++)
        {
            checkCudaErrors(hipGetDeviceProperties(&prop[i], i));

            // Only boards based on Fermi or later can support P2P
            if ((prop[i].major >= 2)
#ifdef _WIN32
                // on Windows (64-bit), the Tesla Compute Cluster driver for windows must be enabled to supprot this
                && prop[i].tccDriver
#endif
               )
            {
                // This is an array of P2P capable GPUs
                gpuid[gpu_p2p_count++] = i;
            }
        }

        // Show all the combinations of support P2P GPUs
        int can_access_peer_0_1, can_access_peer_1_0;

        if (gpu_p2p_count >= 2)
        {
            for (int i = 0; i < gpu_p2p_count-1; i++)
            {
                for (int j = 1; j < gpu_p2p_count; j++)
                {
                    checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer_0_1, gpuid[i], gpuid[j]));
                    fprintf(pFile, "> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[i]].name, gpuid[i],
                           prop[gpuid[j]].name, gpuid[j] ,
                           can_access_peer_0_1 ? "Yes" : "No");

                }
            }

            for (int j = 1; j < gpu_p2p_count; j++)
            {
                for (int i = 0; i < gpu_p2p_count-1; i++)
                {
                    checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer_1_0, gpuid[j], gpuid[i]));
                    fprintf(pFile, "> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[j]].name, gpuid[j],
                           prop[gpuid[i]].name, gpuid[i] ,
                           can_access_peer_1_0 ? "Yes" : "No");

						   
                }
            }
        }
    }

    // csv masterlog info
    // *****************************
    // exe and CUDA driver name
    fprintf(pFile, "\n");
		
    std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
    char cTemp[16];

    // driver version
    sProfileString += ", CUDA Driver Version = ";
#ifdef WIN32
    sprintf_s(cTemp, 10, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#else
    sprintf(cTemp, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#endif
    sProfileString +=  cTemp;

    // Runtime version
    sProfileString += ", CUDA Runtime Version = ";
#ifdef WIN32
    sprintf_s(cTemp, 10, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#else
    sprintf(cTemp, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#endif
    sProfileString +=  cTemp;

    // Device count
    sProfileString += ", NumDevs = ";
#ifdef WIN32
    sprintf_s(cTemp, 10, "%d", deviceCount);
#else
    sprintf(cTemp, "%d", deviceCount);
#endif
    sProfileString += cTemp;

    // Print Out all device Names
    for (dev = 0; dev < deviceCount; ++dev)
    {
#ifdef _WIN32
        sprintf_s(cTemp, 13, ", Device%d = ", dev);
#else
        sprintf(cTemp, ", Device%d = ", dev);
#endif
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        sProfileString += cTemp;
        sProfileString += deviceProp.name;
    }

    sProfileString += "\n";
    fprintf(pFile, "%s", sProfileString.c_str());
	

	
	fclose(pFile);

	// read back the text contents of the file
	long f_size;
	char* code;
	size_t code_s, result;
	FILE* fp = fopen("temp.txt", "r");
	fseek(fp, 0, SEEK_END);
	f_size = ftell(fp); /* This returns 29696, but file is 85 bytes */
	fseek(fp, 0, SEEK_SET);
	code_s = sizeof(char) * f_size;
	code = (char *)malloc(code_s);
	result = fread(code, 1, f_size, fp); 
		

	  ret = env->NewStringUTF(code);

   return ret;	
	
}

   
   
